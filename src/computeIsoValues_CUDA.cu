#include "hip/hip_runtime.h"
#include "marchingCubeGrid.h"
#include "computeIsoValues_CUDA.h"

void MarchingCubeGrid::computeIsoValues(const std::vector<glm::vec3> points, double resolution)
{
    double influenceRadius = resolution * 4.0;
    double influenceRadius2 = influenceRadius*influenceRadius;
    double influenceRadius6 = pow(influenceRadius, 6);

    std::vector<double> sumWj;
    std::vector<glm::vec3> sumRjWj;

    int nbGridVertices = getNbVertices();
    sumWj.resize(nbGridVertices, 0.0);
    sumRjWj.resize(nbGridVertices, glm::vec3(0.0,0.0,0.0));

    int nbPoints = points.size();
    for (int p = 0; p < nbPoints; ++p)
    {
        CloudVolume volume;
        volume = getCellsInRadius(points[p], influenceRadius);

        glm::vec3 vertexPos;
        for (int iz=volume.minimum.z; iz<=volume.maximum.z; ++iz)
        {
            for (int iy=volume.minimum.y; iy<=volume.maximum.y; ++iy)
            {
                for (int ix=volume.minimum.x; ix<=volume.maximum.x; ++ix)
                {
                    unsigned int cellIndex = getGridIndex(ix, iy, iz);
                    vertexPos = getVertexPosition(ix, iy, iz);
                    //unsigned int cellIndex = 0;//getGridIndex(ix, iy, iz);
                    //vertexPos = glm::vec3(0.0f,0.0f,0.0f);//getVertexPosition(ix, iy, iz);

                    glm::vec3 delta(vertexPos);
                    delta -= points[p];

                    double dist2 = delta.x*delta.x + delta.y*delta.y + delta.z*delta.z;
                    if (dist2 < influenceRadius2)
                    {
                        double dist = sqrt(dist2);
                        double Wj = pow((1.0 - pow(dist/influenceRadius,2)), 3);

                        glm::vec3 gradWj(delta);
                        gradWj *= -6.0*pow(influenceRadius2-dist2, 2) / influenceRadius6;

                        sumWj[cellIndex] += Wj;

                        sumRjWj[cellIndex].x += points[p].x*Wj;
                        sumRjWj[cellIndex].y += points[p].y*Wj;
                        sumRjWj[cellIndex].z += points[p].z*Wj;
                    }
                }
            }
        }
    }

    glm::vec3 vertexPos;
    for (int c = 0; c < nbGridVertices; ++c)
    {
        unsigned int ix = getIndex(c, 0);
        unsigned int iy = getIndex(c, 1);
        unsigned int iz = getIndex(c, 2);

        double isoValue = 1.0;
        vertexPos = getVertexPosition(ix, iy, iz);

        glm::vec3 averagePosition(sumRjWj[c]);
        averagePosition /= sumWj[c];

        glm::vec3 deltaToAverage(vertexPos);
        deltaToAverage -= averagePosition;

        isoValue = sqrt(deltaToAverage.x*deltaToAverage.x +
                               deltaToAverage.y*deltaToAverage.y +
                               deltaToAverage.z*deltaToAverage.z);
        isoValue -= resolution;
        setScalarValue(ix, iy, iz, isoValue);
    }
}
